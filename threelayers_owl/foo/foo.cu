#include "hip/hip_runtime.h"
#include <iostream>
#include <owl/owl.h>

__global__ void testKernel() {
	printf("foo kernel!\n");
}

__declspec(dllexport) void foo(){
	OWLContext owlContext = owlContextCreate(nullptr, 1);
	testKernel<<<1,1>>>();
	hipDeviceSynchronize();
	owlContextDestroy(owlContext);
	std::cout << "foo done!\n";
}
