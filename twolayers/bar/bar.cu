
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void barKernel() {
	printf("bar!\n");
}

__declspec(dllexport) void bar(){
	barKernel<<<1,1>>>();
	std::cout << "bar done!\n";
}
