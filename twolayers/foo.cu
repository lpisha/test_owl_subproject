
#include <hip/hip_runtime.h>
#include <iostream>

void bar();

__global__ void fooKernel() {
	printf("foo!\n");
}

int main(){
	bar();
	fooKernel<<<1,1>>>();
	hipDeviceSynchronize();
	std::cout << "foo done!\n";
	return 0;
}
