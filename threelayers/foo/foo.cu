
#include <hip/hip_runtime.h>
#include <iostream>

void bar();

__global__ void fooKernel() {
	printf("foo!\n");
}

__declspec(dllexport) void foo(){
	bar();
	fooKernel<<<1,1>>>();
	hipDeviceSynchronize();
	std::cout << "foo done!\n";
}
