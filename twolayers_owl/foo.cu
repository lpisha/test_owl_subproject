#include "hip/hip_runtime.h"
#include <iostream>
#include <owl/owl.h>

__global__ void testKernel() {
	printf("Hello world!\n");
}

/*__declspec(dllexport) void foo()*/
int main(){
	OWLContext owlContext = owlContextCreate(nullptr, 1);
	testKernel<<<1,1>>>();
	hipDeviceSynchronize();
	owlContextDestroy(owlContext);
	std::cout << "foo done!\n";
	return 0;
}
